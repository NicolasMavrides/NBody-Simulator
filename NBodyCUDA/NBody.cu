#include "hip/hip_runtime.h"
#define _CRT_SECURE_NO_WARNINGS
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <ctype.h>
#include "NBody.h"
#include "NBodyVisualiser.h"
#include "hip/hip_runtime.h"
#include ""

#define USER_NAME "aca15npm"		//replace with your username
#define SIZE 256

void print_help();
void step(void);
int numOfBodies;
u_char dimension;
MODE op_mode;
int iterations;
char* filename;
nbody* allBodies;        // nbody array for all simulation bodies
FILE* input_file;
int fileUsed = 0;        // is a CSV file being used? 0 (false) by default
float* heatDensities;   // heat activity map

// device variables/pointers (used as __shared__ variables to store in the GPU cache)
__shared__ nbody* d_allBodies;
__shared__ float* d_heatDensities;

int main(int argc, char* argv[]) {
	// use the current time to seed the random value generator
	srand(time(0));

	// create timers but only 1 will be used depending on op_mode specified
	time_t start = clock();            // create timer for CPU mode
	hipEvent_t start_cuda, stop_cuda;  // and for GPU mode

	// GPU timer requires event records
	hipEventCreate(&start_cuda);
	hipEventCreate(&stop_cuda);
	hipEventRecord(start_cuda);  // start cuda timer

	if (argc < 4 || argc % 2 != 0 || argc > 8) {
		printf("Insufficient or incorrect arguments given to run the program. Please re-run with arguments as follows: \n");
		printf("\n");
		print_help();
		exit(1);
	}

	else if (argc >= 4) {
		// CHECK THAT PARAMS ARE GIVEN SENSIBLE VALUES. EXIT IF NOT.
		if (atoi(argv[1]) < 1) {  // make sure at least 1 body is specified for simulation
			printf("Invalid number of bodies specified. Please make sure you specify at least 1 body.\n");
			exit(1);
		}
		else if (atoi(argv[2]) < 1) {  // make sure dimension is at least 1
			printf("Invalid dimension specified. Please make sure you specify a dimension of at least 1.\n");
			exit(1);
		}
		// make sure OPMODE is either CPU or OpenMP
		if (strcmp(argv[3], "CPU") == 0) {
			op_mode = CPU;
		}
		else if (strcmp(argv[3], "OPENMP") == 0) {
			op_mode = OPENMP;
		}
		else if (strcmp(argv[3], "CUDA") == 0) {
			op_mode = CUDA;
		}
		else {
			printf("Invalid operation mode. Please make sure you specify either CPU or OPENMP as an operation mode.\n");
			exit(1);
		}
		// Once checks pass, assign param values to variable
		numOfBodies = atoi(argv[1]);
		dimension = atoi(argv[2]);
	}


	allBodies = (nbody*)malloc(sizeof(nbody) * numOfBodies);  // allocate host memory for allBodies

	if (argc >= 6) {
		for (int i = 4; i < argc; i += 2) {
			// Loop through the optional arguments, check that flags are correct and that neither is entered twice
			// if an iteration flag exists, get the number of iterations
			if (strcmp(argv[i], "-i") == 0 && strcmp(argv[i], argv[i - 2]) != 0) {
				if (atoi(argv[i + 1]) < 1) {
					printf("Invalid number of iterations specified. Please specify an interation number of at least 1.\n");
					exit(1);
				}
				else {
					iterations = atoi(argv[i + 1]);
				}
			}
			// if a CSV file flag exists, open it and extract the nbodies for the simulation
			else if (strcmp(argv[i], "-f") == 0 && strcmp(argv[i], argv[i - 2]) != 0) {
				filename = argv[i + 1];
				input_file = fopen(filename, "r");

				if (input_file == NULL) {  // check that the file actually exists...
					printf("Specified file not found. Please check that you have entered the directory and/or filename correctly.\n");
					exit(1);
				}
				else {
					fileUsed = 1;			   // set global variable to indicate file is being used
					char fileLine[250];        // allocate reasonable size to read in each file line
					u_int noOfInputBodies = 0; // this will keep track of how many bodies exist in the input file
					while (!feof(input_file)) {
						fgets(fileLine, 250, input_file);
						if (fileLine[0] != '#') {
							char* commaPtr;					 // pointer for each comma
							char value[15];					 // string to store each extracted value
							float value_f;					 // value converted to float
							nbody body = { 0, 0, 0, 0, 0 };  // create new body for each line read in

							// init variables for finding number values
							char from = 0;
							char to = 0;
							commaPtr = strchr(fileLine, ',');

							for (int i = 0; i < 5; i++) {  // file should contain 4 commas so iterate until the 4th comma is located

								// if it's already found all 4 commas then look for the line terminator instead
								if (commaPtr == NULL) {
									to = strchr(fileLine, '\0') - fileLine;
								}
								else {
									to = strchr(commaPtr, ',') - fileLine;
								}

								// calculate to/from indexes of next value between commas
								strncpy(value, (fileLine + from), (to - from));
								value[to - from] = '\0';

								// if 2 commas are next to each other or there is no value, generate a default value
								if (to - from < 2) {
									// check which field the value belongs to and assign default value accordingly
									if (i == 0 || i == 1) {
										value_f = (float)rand() / (float)(RAND_MAX);  // starting positions
									}
									else if (i == 2 || i == 3) {  // velocity components
										value_f = 0;
									}
									else if (i == 4) {  // mass
										value_f = 1 / numOfBodies;
									}
								}
								else {
									value_f = atof(value);  // else take the string value from file and convert to float
								}

								// finally, move on to the next comma if not already on the last one
								if (commaPtr != NULL) {
									commaPtr += 1;
									from = commaPtr - fileLine;
									commaPtr = strchr(commaPtr, ',');
								}

								// assign values to the body fields depending on which field it's on
								if (i == 0) {	   // x start position
									body.x = value_f;
								}
								else if (i == 1) { // y start position
									body.y = value_f;
								}
								else if (i == 2) { // x velocity
									body.vx = value_f;
								}
								else if (i == 3) { // y velocity
									body.vy = value_f;
								}
								else if (i == 4) { // mass
									body.m = value_f;
								}
							}
							allBodies[noOfInputBodies] = body;  // add body to the array of nbodies
							// and increment body count to keep track of how many are extracted from the CSV file
							noOfInputBodies += 1;
						}
					}

					fclose(input_file);  // close file once no longer needed

					if (numOfBodies != noOfInputBodies) {
						printf("The number of bodies you have specified does not match the number of bodies in the input file. Please check your arguments and try again.\n");
						exit(1);
					}
				}
			}

			else if (strcmp(argv[i], argv[i - 2]) == 0) {
				printf("It appears you have entered an optional argument flag twice. Please re-run with arguments as follows: \n");
				printf("\n");
				print_help();
				exit(1);
			}

			else { // else advise the user that something is wrong
				printf("There appears to be an issue with the arguments you have given, such as not providing the correct flags. Please re-run with arguments as follows: \n");
				printf("\n");
				print_help();
				exit(1);
			}
		}
	}

	if (fileUsed == 0) {  // if CSV file wasn't used, generate bodies with random values
		for (int i = 0; i < numOfBodies; i++) {
			// Generate a set of random data for bodies
			float x = (float)rand() / (float)(RAND_MAX);
			float y = (float)rand() / (float)(RAND_MAX);
			float vx = 0.0f;
			float vy = 0.0f;
			float m = (float)1 / (float)numOfBodies;

			// create body object with values generated
			nbody body = { x, y, vx, vy, m };
			allBodies[i] = body;  // assign body to next position in body array
		}
	}

	// Start simulation here
	// If in CUDA mode, then the device variable for allBodies on the GPU needs to be setup. Otherwise this part can be just skipped
	if (op_mode == CUDA) {
		// copy over from host variable
		hipMalloc((void**)&d_allBodies, (sizeof(nbody) * numOfBodies));
		hipMemcpy(d_allBodies, allBodies, (sizeof(nbody) * numOfBodies), hipMemcpyHostToDevice);
	}

	// start visualiser if no iteration argument was provided
	if (iterations == 0) {
		heatDensities = (float*)malloc(sizeof(float) * dimension * dimension);

		if (op_mode == CUDA) {
			// create heatDensities variable for device and copy over from host variable
			hipMalloc((void**)&d_heatDensities, (sizeof(float) * dimension * dimension));
			hipMemcpy(d_heatDensities, heatDensities, (sizeof(float) * dimension * dimension), hipMemcpyHostToDevice);

			// start visualiser with CUDA mode
			initViewer(numOfBodies, dimension, op_mode, step_cuda);
			setNBodyPositions(d_allBodies);
			setHistogramData(d_heatDensities);
			startVisualisationLoop();
			hipFree(d_heatDensities);  // free memory on GPU used for device heatmap
		}
		else {
			// start visualiser in non-CUDA mode
			initViewer(numOfBodies, dimension, op_mode, step);
			setNBodyPositions(allBodies);
			setHistogramData(heatDensities);
			startVisualisationLoop();
		}

		free(heatDensities);  // free memory used for host heatmap
	}
	else {
		// if a number of iterations are specified then visualiser is not needed. Call only step function
		if (op_mode == CUDA) {
			// if in CUDA mode, configure the grid of thread blocks and run the kernel
			for (int c = 0; c < iterations; c++) {
				step_cuda();
			}
			hipFree(d_allBodies);  // free memory on GPU used for device bodies
		}
		else {
			// otherwise run the step() function for the number of iterations given
			for (int c = 0; c < iterations; c++) {
				step();
			}
		}

	}

	free(allBodies);  // free memory used for allBodies

	time_t end = clock(); // end timer
	// get time in milliseconds and print out execution time
	u_int time_spent = (end - start) * 1000 / CLOCKS_PER_SEC;
	printf("Execution time %d seconds and %d milliseconds \n", (time_spent / 1000), (time_spent % 1000));

	return 0;
}


void step_cuda(void) {
	// Perform the main simulation of the NBody system per iteration
	dim3 threadsPerBlock(SIZE);
	dim3 blocksPerGrid(numOfBodies / SIZE + 1);
	UpdateValues_Cuda << <blocksPerGrid, threadsPerBlock >> > (d_allBodies, numOfBodies);

	if (iterations == 0) {
		hipMemset(d_heatDensities, 0.0f, (sizeof(float) * dimension * dimension));
		CountHeatDensitiesCuda << <blocksPerGrid, threadsPerBlock >> > (d_allBodies, d_heatDensities, numOfBodies, dimension);
	}
	hipDeviceSynchronize();
}

__global__ void UpdateValues_Cuda(nbody* d_allBodies, int numOfBodies) {

	int j = (blockIdx.x * blockDim.x) + threadIdx.x;

	float totalForceX = 0;  // total forces acted on x-component of each body
	float totalForceY = 0;  // total forces acted on y-component of each body
	float bodyAccelX = 0;   // calculated x-component acceleration of each body
	float bodyAccelY = 0;   // calculated y-component acceleration of each body

	for (int k = 0; k < numOfBodies; k++) {
		if (k != j) {
			float vec_x = d_allBodies[k].x - d_allBodies[j].x;		// x-position vector
			float vec_y = d_allBodies[k].y - d_allBodies[j].y;		// y-position vector
			float vec_mag = sqrt(vec_x * vec_x + vec_y * vec_y);    // vector magnitude
			float denominator = (vec_mag * vec_mag + SOFTENING * SOFTENING);

			// calculate force components
			float x_comp = (d_allBodies[k].m * vec_x) / sqrt(denominator * denominator * denominator);
			float y_comp = (d_allBodies[k].m * vec_y) / sqrt(denominator * denominator * denominator);

			// update force components
			totalForceX += x_comp;
			totalForceY += y_comp;
		}
	}

	// calculate acceleration components by multiplying the total forces by G
	bodyAccelX = totalForceX * G;
	bodyAccelY = totalForceY * G;

	// calculate velocity components
	d_allBodies[j].vx += dt * bodyAccelX;
	d_allBodies[j].vy += dt * bodyAccelY;

	// calculate new body position components
	d_allBodies[j].x += dt * d_allBodies[j].vx;
	d_allBodies[j].y += dt * d_allBodies[j].vy;
}


__global__ void CountHeatDensitiesCuda(nbody* d_allBodies, float* d_heatDensities, int numOfBodies, int dimension) {
	// calculate and update body densities on each grid during simulation
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	float x_normalised = floor(dimension * d_allBodies[i].x);
	float y_normalised = floor(dimension * d_allBodies[i].y);

	// get the xy_normalised value from the x and y values above
	int xy_normalised = x_normalised + (y_normalised * dimension);  // multiply y_normalised by D as y is the row value

	// make sure we ignore bodies that have gone out of range of the grid
	if ((xy_normalised >= 0) && (xy_normalised < dimension * dimension) && (i < numOfBodies)) {
		atomicAdd(&d_heatDensities[xy_normalised], ((float)dimension / numOfBodies));
	}
}


void step(void)
{
	// Perform the main simulation of the NBody system per iteration
	int j, k;
#pragma omp parallel for private(j, k) schedule(dynamic) if (op_mode == OPENMP)
	for (j = 0; j < numOfBodies; j++) {
		float totalForceX = 0;  // total forces acted on x-component of each body
		float totalForceY = 0;  // total forces acted on y-component of each body
		float bodyAccelX = 0;    // calculated x-component acceleration of each body
		float bodyAccelY = 0;    // calculated y-component acceleration of each body

		for (int k = 0; k < numOfBodies; k++) {
			if (k != j) {
				totalForceX += SumForces(allBodies[j], allBodies[k]).x;
				totalForceY += SumForces(allBodies[j], allBodies[k]).y;
			}
		}

		// calculate acceleration components by multiplying the total forces by G
		bodyAccelX = totalForceX * G;
		bodyAccelY = totalForceY * G;

		// calculate velocity components
		allBodies[j].vx += dt * bodyAccelX;
		allBodies[j].vy += dt * bodyAccelY;

		// calculate new body position components
		allBodies[j].x += dt * allBodies[j].vx;
		allBodies[j].y += dt * allBodies[j].vy;
		//PrintBodyInfo(allBodies[j]);
	}

	// make heatmap calculations if using visualiser mode
	if (iterations == 0) {
		CountHeatDensities();
	}
}

// Calculation of force on a body i by a body j
force SumForces(nbody i, nbody j) {

	float vec_x = j.x - i.x;						        // x-position vector
	float vec_y = j.y - i.y;						        // y-position vector
	float vec_mag = sqrt(vec_x * vec_x + vec_y * vec_y);    // vector magnitude
	float denominator = (vec_mag * vec_mag + SOFTENING * SOFTENING);
	// calculate force components
	float x_comp = (j.m * vec_x) / sqrt(denominator * denominator * denominator);
	float y_comp = (j.m * vec_y) / sqrt(denominator * denominator * denominator);

	// store force components in structure which is returned
	struct force force;
	force.x = x_comp;
	force.y = y_comp;

	return force;

}

void CountHeatDensities(void) {
	// calculate and update body densities on each grid during simulation
	int i;
	#pragma omp parallel for private(i) shared(heatDensities) if (op_mode == OPENMP)
	for (i = 0; i < numOfBodies; i++) {
		float x_normalised = floor(dimension * allBodies[i].x);
		float y_normalised = floor(dimension * allBodies[i].y);
		// get the xy_normalised value from the x and y values above
		int xy_normalised = x_normalised + (y_normalised * dimension);  // multiply y_normalised by D as y is the row value

																		// make sure we ignore bodies that have gone out of range of the grid
		if ((xy_normalised >= 0) && (xy_normalised < dimension * dimension)) {
			#pragma omp atomic
			heatDensities[xy_normalised]++;
		}
	}

	int x;
	#pragma omp parallel for private(x) shared(heatDensities) if (op_mode == OPENMP)
	for (x = 0; x < (dimension * dimension); x++) {
		#pragma omp atomic
		heatDensities[x] /= numOfBodies;
	}
}

void print_help() {
	printf("nbody_%s N D M [-i I] [-i input_file]\n", USER_NAME);
	printf("where:\n");
	printf("\tN                Is the number of bodies to simulate.\n");
	printf("\tD                Is the integer dimension of the activity grid. The Grid has D*D locations.\n");
	printf("\tM                Is the operation mode, either  'CPU' or 'OPENMP'\n");
	printf("\t[-i I]           Optionally specifies the number of simulation iterations 'I' to perform. Specifying no value will use visualisation mode. \n");
	printf("\t[-f input_file]  Optionally specifies an input file with an initial N bodies of data. If not specified random data will be created.\n");
}